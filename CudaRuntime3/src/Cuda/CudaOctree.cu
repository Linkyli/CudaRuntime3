#include "hip/hip_runtime.h"
﻿#include<../src/Cuda/Octree_DeeperCut.cuh>

using namespace std;



CudaOctree AllocMemoryForCudaOctreeForThreeLevel(Point3f& origin, Point3f& halfDimension) {

	OctreeNode* Root;
	Node* OctreeHostBuffer;
	Node* OctrerDevieBuffer;//Device端数组
	BitBricks* HostBrick;
	BitBricks* DeviceBrick;
	Cutter* HostCutter;
	Cutter* DeviceCutter;

	Root = (OctreeNode*)malloc(8 * sizeof(OctreeNode));
	OctreeHostBuffer = (Node*)malloc(8 * BrickLength * sizeof(Node));
	HostBrick = (BitBricks*)malloc(8 * BrickLength * sizeof(BitBricks));
	HostCutter = (Cutter*)malloc(sizeof(Cutter));

	hipMalloc((void**)&(OctrerDevieBuffer), 8 * BrickLength * sizeof(Node));
	hipMalloc((void**)&(DeviceBrick), 8 * BrickLength * sizeof(BitBricks));
	hipMalloc((void**)&(DeviceCutter), sizeof(Cutter));

	Point3f NodeSubOrigin = Point3f(origin.x - halfDimension.x * 1 / 2,
		origin.y - halfDimension.y * 1 / 2, origin.z - halfDimension.z * 1 / 2);
	Point3f NodeHalfDimension = Point3f(halfDimension.x / 2, halfDimension.y / 2, halfDimension.z / 2);
	for (int i = 0; i < 8; i++) {
		Root[i].Nodes = &OctreeHostBuffer[i * BrickLength];
		Root[i].Origin.x = NodeSubOrigin.x + (i & 4) / 2 * NodeHalfDimension.x;
		Root[i].Origin.y = NodeSubOrigin.y + (i & 2) * NodeHalfDimension.y;
		Root[i].Origin.z = NodeSubOrigin.z + (i & 1) * 2 * NodeHalfDimension.z;
		Root[i].HalfDimension = NodeHalfDimension;
	}

	Point3f SubOrigin = Point3f(origin.x - halfDimension.x * 15 / 16,
		origin.y - halfDimension.y * 15 / 16, origin.z - halfDimension.z * 15 / 16);

	Point3f HalfDimension = Point3f(halfDimension.x / 32, halfDimension.y / 32, halfDimension.z / 32);//每个Node的尺寸
	HostCutter->SubOrigin = SubOrigin;
	HostCutter->halfDimension = HalfDimension;

	return CudaOctree(OctreeHostBuffer, OctrerDevieBuffer, HostBrick, DeviceBrick, HostCutter, DeviceCutter, Root);
}

CudaOctree AllocMemoryForCudaOctree(Point3f& origin, Point3f& halfDimension) {

	Node* OctreeHostBuffer;
	Node* OctrerDevieBuffer;//Device端数组
	BitBricks* HostBrick;
	BitBricks* DeviceBrick;
	Cutter* HostCutter;
	Cutter* DeviceCutter;

	OctreeHostBuffer = (Node*)malloc(BrickLength * sizeof(Node));
	HostBrick = (BitBricks*)malloc(BrickLength * sizeof(BitBricks));
	HostCutter = (Cutter*)malloc(sizeof(Cutter));

	hipMalloc((void**)&(OctrerDevieBuffer), BrickLength * sizeof(Node));
	hipMalloc((void**)&(DeviceBrick), BrickLength * sizeof(BitBricks));
	hipMalloc((void**)&(DeviceCutter), sizeof(Cutter));

	Point3f SubOrigin = Point3f(origin.x - halfDimension.x * 15 / 16,
		origin.y - halfDimension.y * 15 / 16, origin.z - halfDimension.z * 15 / 16);

	Point3f HalfDimension = Point3f(halfDimension.x / 16, halfDimension.y / 16, halfDimension.z / 16);
	HostCutter->SubOrigin = SubOrigin;
	HostCutter->halfDimension = HalfDimension;

	return CudaOctree(OctreeHostBuffer, OctrerDevieBuffer, HostBrick, DeviceBrick, HostCutter, DeviceCutter,nullptr);
}

__device__ bool IsIntersectForCuda(float3& origin, Point3f& halfDimension, Cutter* Device_Cutter) {
	if (abs(origin.x - Device_Cutter->CutterBox.Origin.x) < abs(halfDimension.x + (Device_Cutter->CutterBox.length / 2))
		&& abs(origin.y - Device_Cutter->CutterBox.Origin.y) < abs(halfDimension.y + (Device_Cutter->CutterBox.width / 2))
		&& abs(origin.z - Device_Cutter->CutterBox.Origin.z) < abs(halfDimension.z + (Device_Cutter->CutterBox.height / 2))
		)
	{
		//cout << "相交" << endl
		return true;
	}
	else
	{
		//cout << "不相交" << endl;
		return false;
	}
}

__device__ bool IsIntersectForCudaBrick(float3& origin, float3& halfDimension, Cutter* Device_Cutter) {
	if (abs(origin.x - Device_Cutter->CutterBox.Origin.x) < abs(halfDimension.x + (Device_Cutter->CutterBox.length / 2))
		&& abs(origin.y - Device_Cutter->CutterBox.Origin.y) < abs(halfDimension.y + (Device_Cutter->CutterBox.width / 2))
		&& abs(origin.z - Device_Cutter->CutterBox.Origin.z) < abs(halfDimension.z + (Device_Cutter->CutterBox.height / 2))
		)
	{
		//cout << "相交" << endl
		return true;
	}
	else
	{
		//cout << "不相交" << endl;
		return false;
	}
}
__device__ bool IsInOfCudaNode(float3& OctreeVer, Cutter& Device_Cutter) {
	float X = abs(OctreeVer.x - Device_Cutter.CutterPos.x);
	float Y = abs(OctreeVer.y - Device_Cutter.CutterPos.y);
	float Z = (OctreeVer.z - Device_Cutter.CutterPos.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > Device_Cutter.CutterSize.y || Y > Device_Cutter.CutterSize.y || Z > Device_Cutter.CutterSize.x ||
		(X * X + Y * Y + Z * Z) >= (Device_Cutter.CutterSize.y * Device_Cutter.CutterSize.y))
	{
		return false;
	}

	return true;
}

__device__ float3 GetOctreeVertexForCuda(int number, float3& origin, Point3f& halfDimension) {
	float3 OctreeVertex;

	OctreeVertex.x = origin.x - halfDimension.x;
	OctreeVertex.x = OctreeVertex.x + (number & 4) / 2 * halfDimension.x;

	OctreeVertex.y = origin.y - halfDimension.y;
	OctreeVertex.y = OctreeVertex.y + (number & 2) * halfDimension.y;

	OctreeVertex.z = origin.z - halfDimension.z;
	OctreeVertex.z = OctreeVertex.z + (number & 1) * 2 * halfDimension.z;

	return OctreeVertex;
}

__device__ float3 GetBrickVertexForCuda(int number, float3& origin, float3& halfDimension) {
	float3 OctreeVertex;

	OctreeVertex.x = origin.x - halfDimension.x;
	OctreeVertex.x = OctreeVertex.x + (number & 4) / 2 * halfDimension.x;

	OctreeVertex.y = origin.y - halfDimension.y;
	OctreeVertex.y = OctreeVertex.y + (number & 2) * halfDimension.y;

	OctreeVertex.z = origin.z - halfDimension.z;
	OctreeVertex.z = OctreeVertex.z + (number & 1) * 2 * halfDimension.z;

	return OctreeVertex;
}

__global__ void DeeperCutForCuda(float3 SubOrigin, float3 halfDimension, int NodeIndex, BitBricks* DeviceBricks, Cutter* Device_Cutter) {
	
	//int index = threadIdx.x ;
	int RealIndex;
	float3 SubTemp;
	int Forward;

	BYTE mask;
	//BYTE temp = DeviceBricks[NodeIndex].brick[BrickIndex];
	//mask = mask >> 4;
	//printf("%d号的index：%d\n", NodeIndex, RealIndex);
	for (int i = 0; i < 8;i++) {
	
		RealIndex = threadIdx.x * 8 + i;
		Forward =RealIndex % 8;
		mask = 1;//使用或|,当所在bit位置为1时则代表已经被切削
		mask = mask << i;

		SubTemp.x = SubOrigin.x + (RealIndex & 15) * halfDimension.x * 2;
		SubTemp.y = SubOrigin.y + (RealIndex & 240) / 16 * halfDimension.y * 2;
		SubTemp.z = SubOrigin.z + (RealIndex & 3840) / 256 * halfDimension.z * 2;

	   if (IsInOfCudaNode(SubTemp, *Device_Cutter)) {
		   DeviceBricks[NodeIndex].brick[threadIdx.x] = (DeviceBricks[NodeIndex].brick[threadIdx.x] | mask);
	   }
	}

}

__global__ void DeeperCutForCuda2(float3 SubOrigin, float3 halfDimension, int NodeIndex, BitBricks* DeviceBricks, Cutter* Device_Cutter) {

	int index = blockIdx.x * 256 + threadIdx.x;
	int RealIndex;
	float3 SubTemp;
	int Forward;

	BYTE mask;
	//BYTE temp = DeviceBricks[NodeIndex].brick[BrickIndex];
	//mask = mask >> 4;
	//printf("%d号的index：%d\n", NodeIndex, RealIndex);
	for (int i = 0; i < 8; i++) {

		RealIndex = index * 8 + i;
		Forward = RealIndex % 8;
		mask = 1;//使用或|,当所在bit位置为1时则代表已经被切削
		mask = mask << i;

		SubTemp.x = SubOrigin.x + (RealIndex & 31) * halfDimension.x * 2;
		SubTemp.y = SubOrigin.y + (RealIndex & 992) / 32 * halfDimension.y * 2;
		SubTemp.z = SubOrigin.z + (RealIndex & 31744) / 1024 * halfDimension.z * 2;

		if (IsInOfCudaNode(SubTemp, *Device_Cutter)) {
			DeviceBricks[NodeIndex].brick[index] = (DeviceBricks[NodeIndex].brick[index] | mask);
		}
	}

}


__global__  void DynamicCut(Node* OctreeDeviceBuffer, BitBricks* DeviceBricks, Cutter* CurrCutter,Point3f& SubOrigin) {

	int index = blockIdx.x * 256 + threadIdx.x;
	
	if (OctreeDeviceBuffer[index].exist) return;//true代表已经被切削掉

	float3 OctreeVertex;
	float3 SubTemp;
	SubTemp.x = SubOrigin.x + (index & 15) * CurrCutter->halfDimension.x * 2;        //CurrCutter->
	SubTemp.y = SubOrigin.y + (index & 240) / 16 * CurrCutter->halfDimension.x * 2;  //CurrCutter->
	SubTemp.z = SubOrigin.z + (index & 3840) / 256 * CurrCutter->halfDimension.x * 2;//CurrCutter->

	if (IsIntersectForCuda(SubTemp, CurrCutter->halfDimension, CurrCutter)) {
		//printf("相交:\n");
		for (int i = 0; i <= 7; ++i) {
			OctreeVertex = GetOctreeVertexForCuda(i, SubTemp, CurrCutter->halfDimension);
			if (IsInOfCudaNode(OctreeVertex, *CurrCutter)) {
				if (i == 7) {
					OctreeDeviceBuffer[index].exist = true;
					return;
				}
				continue;
			}
			else {//创建线程执行更深层次的切削
				OctreeDeviceBuffer[index].Sub = true;
				float3 BrickOrigin;
				float3 BrickHalfDimension;

				BrickHalfDimension.x = CurrCutter->halfDimension.x / 16;
				BrickHalfDimension.y = CurrCutter->halfDimension.y / 16;
				BrickHalfDimension.z = CurrCutter->halfDimension.z / 16;

				BrickOrigin.x = SubTemp.x - CurrCutter->halfDimension.x * 15 / 16;
				BrickOrigin.y = SubTemp.y - CurrCutter->halfDimension.y * 15 / 16;
				BrickOrigin.z = SubTemp.z - CurrCutter->halfDimension.z * 15 / 16;
				dim3 dimGrid = (1);
				dim3 dimBlock = (512);
				//printf("调用深度切削");
				DeeperCutForCuda << <dimGrid, dimBlock >> > (BrickOrigin, BrickHalfDimension, index, DeviceBricks, CurrCutter);
				break;
			}
		}
		//if (IsInOfBrick(SubTemp, Device_Cutter)) {D_buffer[threadIdx.x] = true;}
		//else {bri->brick[k] = true;}
	}

}

__global__  void DynamicCut2(Node* OctreeDeviceBuffer, BitBricks* DeviceBricks, Cutter* CurrCutter, Point3f& SubOrigin) {

	int index = blockIdx.x * 256 + threadIdx.x;

	if (OctreeDeviceBuffer[index].exist) return;//true代表已经被切削掉

	float3 OctreeVertex;
	float3 SubTemp;
	SubTemp.x = CurrCutter->SubOrigin.x + (index & 15) * CurrCutter->halfDimension.x * 2;        //CurrCutter->
	SubTemp.y = CurrCutter->SubOrigin.y + (index & 240) / 16 * CurrCutter->halfDimension.x * 2;  //CurrCutter->
	SubTemp.z = CurrCutter->SubOrigin.z + (index & 3840) / 256 * CurrCutter->halfDimension.x * 2;//CurrCutter->

	if (IsIntersectForCuda(SubTemp, CurrCutter->halfDimension, CurrCutter)) {
		//printf("相交:\n");
		for (int i = 0; i <= 7; ++i) {
			OctreeVertex = GetOctreeVertexForCuda(i, SubTemp, CurrCutter->halfDimension);
			if (IsInOfCudaNode(OctreeVertex, *CurrCutter)) {
				if (i == 7) {
					OctreeDeviceBuffer[index].exist = true;
					return;
				}
				continue;
			}
			else {//创建线程执行更深层次的切削
				OctreeDeviceBuffer[index].Sub = true;
				float3 BrickOrigin;
				float3 BrickHalfDimension;

				BrickHalfDimension.x = CurrCutter->halfDimension.x / 32;
				BrickHalfDimension.y = CurrCutter->halfDimension.y / 32;
				BrickHalfDimension.z = CurrCutter->halfDimension.z / 32;

				BrickOrigin.x = SubTemp.x - CurrCutter->halfDimension.x * 31 / 32;
				BrickOrigin.y = SubTemp.y - CurrCutter->halfDimension.y * 31 / 32;
				BrickOrigin.z = SubTemp.z - CurrCutter->halfDimension.z * 31 / 32;
				dim3 dimGrid = (16);
				dim3 dimBlock = (256);
				//printf("调用深度切削");
				DeeperCutForCuda2 << <dimGrid, dimBlock >> > (BrickOrigin, BrickHalfDimension, index, DeviceBricks, CurrCutter);
				break;
			}
		}
		//if (IsInOfBrick(SubTemp, Device_Cutter)) {D_buffer[threadIdx.x] = true;}
		//else {bri->brick[k] = true;}
	}

}



Point3f GetOctreeVertexForNode(int i,int number, CudaOctree& CudaBuffer) {
	Point3f OctreeVertex;

	OctreeVertex.x = CudaBuffer.HostOctree[i].Origin.x - CudaBuffer.HostOctree[i].HalfDimension.x;
	OctreeVertex.x = OctreeVertex.x + (number & 4) / 2 * CudaBuffer.HostOctree[i].HalfDimension.x;

	OctreeVertex.y = CudaBuffer.HostOctree[i].Origin.y - CudaBuffer.HostOctree[i].HalfDimension.y;
	OctreeVertex.y = OctreeVertex.y + (number & 2) * CudaBuffer.HostOctree[i].HalfDimension.y;

	OctreeVertex.z = CudaBuffer.HostOctree[i].Origin.z - CudaBuffer.HostOctree[i].HalfDimension.z;
	OctreeVertex.z = OctreeVertex.z + (number & 1) * 2 * CudaBuffer.HostOctree[i].HalfDimension.z;

	return OctreeVertex;
}

bool IsIntersectForNode(int i,CudaOctree& CudaBuffer, BoundBox& Cutter) {
	if (std::abs(CudaBuffer.HostOctree[i].Origin.x - Cutter.Origin.x) < std::abs(CudaBuffer.HostOctree[i].HalfDimension.x + (Cutter.length / 2))
		&& std::abs(CudaBuffer.HostOctree[i].Origin.y - Cutter.Origin.y) < std::abs(CudaBuffer.HostOctree[i].HalfDimension.y + (Cutter.width / 2))
		&& std::abs(CudaBuffer.HostOctree[i].Origin.z - Cutter.Origin.z) < std::abs(CudaBuffer.HostOctree[i].HalfDimension.z + (Cutter.height / 2))
		)
	{
		//cout << "相交" << endl;
		return true;
	}
	else
	{
		//cout << "不相交" << endl;
		return false;
	}

}
void PrepareCut(CudaOctree& CudaBuffer, Point3f& CutterPos, Point3f& CutterSize, BoundBox& CutterBox) {
	
	//初始化刀具等信息
	CudaBuffer.HostCutter->CutterBox = CutterBox;
	CudaBuffer.HostCutter->CutterPos = CutterPos;
	CudaBuffer.HostCutter->CutterSize = CutterSize;
	hipMemcpy(CudaBuffer.DeviceCutter, CudaBuffer.HostCutter, sizeof(Cutter), hipMemcpyHostToDevice);
	
	//创建4096个线程
	dim3 dimGrid = (16);
	dim3 dimBlock = (256);
	DynamicCut2 << <dimGrid, dimBlock >> > (CudaBuffer.DeviceBuffer, CudaBuffer.DeviceBrick, CudaBuffer.DeviceCutter, CudaBuffer.DeviceCutter->SubOrigin);
	/*
	hipDeviceSynchronize();
	//将Node数组的信息复制返回给Host端
	hipMemcpy(CudaBuffer.HostBuffer, CudaBuffer.DeviceBuffer, BrickLength * sizeof(Node), hipMemcpyDeviceToHost);
	//将brick数组的信息复制返回给host端
	hipMemcpy(CudaBuffer.HostBrick, CudaBuffer.DeviceBrick, BrickLength * sizeof(BitBricks), hipMemcpyDeviceToHost);
	*/
}

void PrepareCutForThreeLevel(CudaOctree& CudaBuffer, Point3f& CutterPos, Point3f& CutterSize, BoundBox& CutterBox) {

	//初始化刀具等信息
	CudaBuffer.HostCutter->CutterBox = CutterBox;
	CudaBuffer.HostCutter->CutterPos = CutterPos;
	CudaBuffer.HostCutter->CutterSize = CutterSize;
	hipMemcpy(CudaBuffer.DeviceCutter, CudaBuffer.HostCutter, sizeof(Cutter), hipMemcpyHostToDevice);

	//创建4096个线程
	dim3 dimGrid = (16);
	dim3 dimBlock = (256);

	Point3f OctreeVertex;
	
	for (int i = 0; i < 8; i++) {
		if (IsIntersectForNode(i, CudaBuffer, CutterBox)) {
			DynamicCut<<<dimGrid,dimBlock>>>(&CudaBuffer.DeviceBuffer[i * 4096],& CudaBuffer.DeviceBrick[i * 4096], CudaBuffer.DeviceCutter, CudaBuffer.HostOctree[i].Origin);
		}
	}


	
	/*
	hipDeviceSynchronize();
	//将Node数组的信息复制返回给Host端
	hipMemcpy(CudaBuffer.HostBuffer, CudaBuffer.DeviceBuffer, BrickLength * sizeof(Node), hipMemcpyDeviceToHost);
	//将brick数组的信息复制返回给host端
	hipMemcpy(CudaBuffer.HostBrick, CudaBuffer.DeviceBrick, BrickLength * sizeof(BitBricks), hipMemcpyDeviceToHost);
	*/
}


/*
if (IsIntersectForCudaBrick(SubTemp, halfDimension, Device_Cutter)) {
	for (int i = 0; i < 8; i++) {
		OctreeVertex = GetBrickVertexForCuda(i, SubTemp, halfDimension);
		if (IsInOfCudaNode(SubTemp, *Device_Cutter)) {
			if (i == 7) {
				DeviceBricks[NodeIndex].brick[BrickIndex] = DeviceBricks[NodeIndex].brick[BrickIndex] | mask;
			}
		}
		else {
			break;
		}
	}

}
*/

