#include "hip/hip_runtime.h"
﻿#include<../src/Cuda/Octree_DeeperCut.cuh>
#include<../src/Octree/Octree.h>
//#include<../src/Octree/OpenGL_Render_Octree.h>

using namespace std;

Device AllocMemory(const int BrickLength) {
	bool* D_buffer;//Device端数组
	hipMalloc((void**)&(D_buffer), BrickLength * sizeof(bool));
	Cutter* Device_Cutter;
	hipMalloc((void**)&(Device_Cutter), sizeof(Cutter));
	return Device(D_buffer, Device_Cutter);
	
}


__device__ bool IsIntersectForBrick(float3& origin, Cutter& Device_Cutter) {
	if (abs(origin.x - Device_Cutter.CutterBox.Origin.x) < abs(Device_Cutter.halfDimension.x + (Device_Cutter.CutterBox.length / 2))
		&& abs(origin.y - Device_Cutter.CutterBox.Origin.y) < abs(Device_Cutter.halfDimension.y + (Device_Cutter.CutterBox.width / 2))
		&& abs(origin.z - Device_Cutter.CutterBox.Origin.z) < abs(Device_Cutter.halfDimension.z + (Device_Cutter.CutterBox.height / 2))
		)
	{
		//cout << "相交" << endl;
		return true;
	}
	else
	{
		//cout << "不相交" << endl;
		return false;
	}
}

__device__ bool IsInOfBrick(float3& OctreeVer, Cutter& Device_Cutter) {
	float X = abs(OctreeVer.x - Device_Cutter.CutterPos.x);
	float Y = abs(OctreeVer.y - Device_Cutter.CutterPos.y);
	float Z = (OctreeVer.z - Device_Cutter.CutterPos.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > Device_Cutter.CutterSize.y || Y > Device_Cutter.CutterSize.y || Z > Device_Cutter.CutterSize.x ||
		(X * X + Y * Y + Z * Z) >= (Device_Cutter.CutterSize.y * Device_Cutter.CutterSize.y))
	{
		return false;
	}

	return true;
}

__device__ float3 GetOctreeVertexForBrick(int number, float3& origin, Point3f& halfDimension) {
	float3 OctreeVertex;

	OctreeVertex.x = origin.x - halfDimension.x;
	OctreeVertex.x = OctreeVertex.x + (number & 4) / 2 * halfDimension.x;

	OctreeVertex.y = origin.y - halfDimension.y;
	OctreeVertex.y = OctreeVertex.y + (number & 2) * halfDimension.y;

	OctreeVertex.z = origin.z - halfDimension.z;
	OctreeVertex.z = OctreeVertex.z + (number & 1) * 2 * halfDimension.z;

	return OctreeVertex;
}


__global__  void Device_Deeper_cut(bool* D_buffer, Cutter& Device_Cutter) {
	if (D_buffer[threadIdx.x]) return;

	float3 SubTemp;
	float3 OctreeVertex;
	SubTemp.x = Device_Cutter.SubOrigin.x;
	SubTemp.y = Device_Cutter.SubOrigin.y;
	SubTemp.z = Device_Cutter.SubOrigin.z;

	SubTemp.x = SubTemp.x + (threadIdx.x & 3) * Device_Cutter.halfDimension.x * 2;
	SubTemp.y = SubTemp.y + (threadIdx.x & 12) / 4 * Device_Cutter.halfDimension.y * 2;
	SubTemp.z = SubTemp.z + (threadIdx.x & 48) / 16 * Device_Cutter.halfDimension.z * 2;

	if (IsIntersectForBrick(SubTemp, Device_Cutter)) {
		//printf("相交:\n");
		for (int i = 0; i <= 7; ++i) {
			OctreeVertex = GetOctreeVertexForBrick(i, SubTemp, Device_Cutter.halfDimension);
			if (IsInOfBrick(OctreeVertex, Device_Cutter)) {
				if (i == 7) {
					D_buffer[threadIdx.x] = true;
					break;
				}
				continue;
			}
		}
		//if (IsInOfBrick(SubTemp, Device_Cutter)) {D_buffer[threadIdx.x] = true;}
		//else {bri->brick[k] = true;}
	}
}

__global__  void Device_Deeper_cut512(bool* D_buffer, Cutter& Device_Cutter) {
	if (D_buffer[threadIdx.x]) return;

	float3 SubTemp;
	SubTemp.x = Device_Cutter.SubOrigin.x + (threadIdx.x & 7) * Device_Cutter.halfDimension.x * 2;
	SubTemp.y = Device_Cutter.SubOrigin.y + (threadIdx.x & 56) / 8 * Device_Cutter.halfDimension.y * 2;
	SubTemp.z = Device_Cutter.SubOrigin.z + (threadIdx.x & 448) / 64 * Device_Cutter.halfDimension.z * 2;
	/*
	SubTemp.x = SubTemp.x + (threadIdx.x & 7) * Device_Cutter.halfDimension.x * 2;
	SubTemp.y = SubTemp.y + (threadIdx.x & 56) / 8 * Device_Cutter.halfDimension.y * 2;
	SubTemp.z = SubTemp.z + (threadIdx.x & 448) / 64 * Device_Cutter.halfDimension.z * 2;
	*/
	if (IsIntersectForBrick(SubTemp, Device_Cutter)) {
		//printf("相交:\n");
		if (IsInOfBrick(SubTemp, Device_Cutter)) {
			D_buffer[threadIdx.x] = true;
		}
		//else {bri->brick[k] = true;}
	}
}

__global__  void Device_Deeper_cut4096(bool* D_buffer, Cutter& Device_Cutter) {
	int index = blockIdx.x * 256 + threadIdx.x;
	if (D_buffer[index]) return;

	float3 SubTemp;
	SubTemp.x = Device_Cutter.SubOrigin.x + (index & 15) * Device_Cutter.halfDimension.x * 2;
	SubTemp.y = Device_Cutter.SubOrigin.y + (index & 240) / 16 * Device_Cutter.halfDimension.y * 2;
	SubTemp.z = Device_Cutter.SubOrigin.z + (index & 3840) / 256 * Device_Cutter.halfDimension.z * 2;
	/*
	SubTemp.x = SubTemp.x + (blockIdx.x) * Device_Cutter.halfDimension.x * 2;
	SubTemp.y = SubTemp.y + (threadIdx.x & 15) * Device_Cutter.halfDimension.y * 2;
	SubTemp.z = SubTemp.z + (threadIdx.x & 240) / 16 * Device_Cutter.halfDimension.z * 2;

	//printf("index: %d SubTemp(%d,%d,%d)\n", index, blockIdx.x, (threadIdx.x & 15), (threadIdx.x & 240) / 16);
	
	SubTemp.x = SubTemp.x + (index & 15) * Device_Cutter.halfDimension.x * 2;
	SubTemp.y = SubTemp.y + (index & 240) / 16 * Device_Cutter.halfDimension.y * 2;
	SubTemp.z = SubTemp.z + (index & 3840) / 256 * Device_Cutter.halfDimension.z * 2;
	*/
	if (IsIntersectForBrick(SubTemp, Device_Cutter)) {
		//printf("相交:\n");
		if (IsInOfBrick(SubTemp, Device_Cutter)) {
			D_buffer[index] = true;
		}
		//else {bri->brick[k] = true;}
	}
}




void Init_Data(Octree* curr, BoundBox& CutterBox, Point3f& CutterPos, Point3f& CutterSize, Device& MyDevice) {

	Cutter Host_Cutter;
	const int Length = 64;

	Host_Cutter.SubOrigin = Point3f(curr->origin.x - curr->halfDimension.x * 3 / 4,
		curr->origin.y - curr->halfDimension.y * 3 / 4, curr->origin.z - curr->halfDimension.z * 3 / 4);
	Host_Cutter.halfDimension = Point3f(curr->halfDimension.x / 4, curr->halfDimension.y / 4, curr->halfDimension.z / 4);
	Host_Cutter.CutterSize = CutterSize;
	Host_Cutter.CutterPos = CutterPos;
	Host_Cutter.CutterBox = CutterBox;

	Bricks* bri = (Bricks*)curr->bricks;
	//bool* D_buffer;//Device端数组
	//hipMalloc((void**)&(D_buffer), Length * sizeof(bool));
	//hipMemcpy(D_buffer, bri->brick, Length * sizeof(bool), hipMemcpyHostToDevice);

	hipMemcpy(MyDevice.D_buffer, bri->brick, Length * sizeof(bool), hipMemcpyHostToDevice);

	//Cutter* Device_Cutter;
	//hipMalloc((void**)&(Device_Cutter), sizeof(Cutter));
	//hipMemcpy(Device_Cutter, &Host_Cutter, sizeof(Cutter), hipMemcpyHostToDevice);
	hipMemcpy(MyDevice.Device_Cutter, &Host_Cutter, sizeof(Cutter), hipMemcpyHostToDevice);

	Device_Deeper_cut << <1, Length >> > (MyDevice.D_buffer, *MyDevice.Device_Cutter);
	hipDeviceSynchronize();
	hipMemcpy(bri->brick, MyDevice.D_buffer, Length * sizeof(bool), hipMemcpyDeviceToHost);

	//hipFree(MyDevice.D_buffer);//释放内存
	//hipFree(MyDevice.Device_Cutter);
}

void Init_Data512(Octree* curr, BoundBox& CutterBox, Point3f& CutterPos, Point3f& CutterSize, Device& MyDevice) {

	Cutter Host_Cutter;
	const int Length = 512;

	Host_Cutter.SubOrigin = Point3f(curr->origin.x - curr->halfDimension.x * 7 / 8,
		curr->origin.y - curr->halfDimension.y * 7 / 8, curr->origin.z - curr->halfDimension.z * 7 / 8);
	Host_Cutter.halfDimension = Point3f(curr->halfDimension.x / 8, curr->halfDimension.y / 8, curr->halfDimension.z / 8);
	Host_Cutter.CutterSize = CutterSize;
	Host_Cutter.CutterPos = CutterPos;
	Host_Cutter.CutterBox = CutterBox;

	Bricks* bri = (Bricks*)curr->bricks;
	bool* D_buffer;//Device端数组
	hipMalloc((void**)&(D_buffer), Length * sizeof(bool));//分配GPU内存
	hipMemcpy(D_buffer, bri->brick, Length * sizeof(bool), hipMemcpyHostToDevice);//为GPU内存赋值

	Cutter* Device_Cutter;
	hipMalloc((void**)&(Device_Cutter), sizeof(Cutter));
	hipMemcpy(Device_Cutter, &Host_Cutter, sizeof(Cutter), hipMemcpyHostToDevice);

	Device_Deeper_cut512 << <1, Length >> > (D_buffer, *Device_Cutter);
	hipDeviceSynchronize();
	hipMemcpy(bri->brick, D_buffer, Length * sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(D_buffer);//释放GPU内存
	hipFree(Device_Cutter);
}

void Init_Data4096(Octree* curr, BoundBox& CutterBox, Point3f& CutterPos, Point3f& CutterSize, Device& MyDevice) {

	Cutter Host_Cutter;

	const int Length = 4096;

	Host_Cutter.SubOrigin = Point3f(curr->origin.x - curr->halfDimension.x * 15 / 16,
		curr->origin.y - curr->halfDimension.y * 15 / 16, curr->origin.z - curr->halfDimension.z * 15 / 16);
	Host_Cutter.halfDimension = Point3f(curr->halfDimension.x / 16, curr->halfDimension.y / 16, curr->halfDimension.z / 16);

	Host_Cutter.CutterSize = CutterSize;
	Host_Cutter.CutterPos = CutterPos;
	Host_Cutter.CutterBox = CutterBox;

	Bricks* bri = (Bricks*)curr->bricks;
	//bool* D_buffer;//Device端数组
	//hipMalloc((void**)&(D_buffer), Length * sizeof(bool));
	hipMemcpy(MyDevice.D_buffer, bri->brick, Length * sizeof(bool), hipMemcpyHostToDevice);

	//Cutter* Device_Cutter;
	//hipMalloc((void**)&(Device_Cutter), sizeof(Cutter));
	hipMemcpy(MyDevice.Device_Cutter, &Host_Cutter, sizeof(Cutter), hipMemcpyHostToDevice);

	dim3 dimGrid = (16);
	dim3 dimBlock = (256);
	//Device_Deeper_cut << <dimGrid, dimBlock >> > (D_buffer, *Device_Cutter);
	Device_Deeper_cut4096 << <dimGrid, dimBlock >> > (MyDevice.D_buffer, *MyDevice.Device_Cutter);
	hipDeviceSynchronize();
	hipMemcpy(bri->brick, MyDevice.D_buffer, Length * sizeof(bool), hipMemcpyDeviceToHost);

	//for (int i = 0; i < 64; i++) {cout << bri->brick[i];}

	//cout << endl;
	//hipFree(D_buffer);
	//hipFree(Device_Cutter);
}



