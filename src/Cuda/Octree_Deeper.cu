#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include"hip/device_functions.h"
#include <iostream>
#include <Windows.h>
#include<stdio.h>
#include<vector>
#include<../src/Octree/Octree.h>
#include<../src/Octree/MyStruct.h>
//#include<../src/Octree/OpenGL_Render_Octree.h>

using namespace std;

__device__ bool IsIntersectForBrick(float3& origin, Cutter& Device_Cutter) {
	if (abs(origin.x - Device_Cutter.CutterBox.Origin.x) < abs(Device_Cutter.halfDimension.x + (Device_Cutter.CutterBox.length / 2))
		&& abs(origin.y - Device_Cutter.CutterBox.Origin.y) < abs(Device_Cutter.halfDimension.y + (Device_Cutter.CutterBox.width / 2))
		&& abs(origin.z - Device_Cutter.CutterBox.Origin.z) < abs(Device_Cutter.halfDimension.z + (Device_Cutter.CutterBox.height / 2))
		)
	{
		//cout << "相交" << endl;
		return true;
	}
	else
	{
		//cout << "不相交" << endl;
		return false;
	}
}

__device__ bool IsInOfBrick(float3& OctreeVer, Cutter& Device_Cutter) {
	float X = abs(OctreeVer.x - Device_Cutter.CutterPos.x);
	float Y = abs(OctreeVer.y - Device_Cutter.CutterPos.y);
	float Z = (OctreeVer.z - Device_Cutter.CutterPos.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > Device_Cutter.CutterSize.y || Y > Device_Cutter.CutterSize.y || Z > Device_Cutter.CutterSize.x ||
		(X * X + Y * Y + Z * Z) >= (Device_Cutter.CutterSize.y * Device_Cutter.CutterSize.y))
	{
		return false;
	}

	return true;
}


__global__  void Device_Deeper_cut(bool* D_buffer, Cutter& Device_Cutter) {
	if (D_buffer[threadIdx.x]) return;

	float3 SubTemp;
	SubTemp.x = Device_Cutter.SubOrigin.x;
	SubTemp.y = Device_Cutter.SubOrigin.y;
	SubTemp.z = Device_Cutter.SubOrigin.z;

	SubTemp.x = SubTemp.x + (threadIdx.x & 3) * Device_Cutter.halfDimension.x / 2;
	SubTemp.y = SubTemp.y + (threadIdx.x & 12) / 4 * Device_Cutter.halfDimension.y / 2;
	SubTemp.z = SubTemp.z + (threadIdx.x & 48) / 16 * Device_Cutter.halfDimension.z / 2;

	if (IsIntersectForBrick(SubTemp, Device_Cutter)) {

		if (IsInOfBrick(SubTemp, Device_Cutter)) {
			D_buffer[threadIdx.x] = true;
		}
		//else {bri->brick[k] = true;}
	}
}



