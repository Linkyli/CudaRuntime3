#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include"hip/device_functions.h"
#include <iostream>
#include <Windows.h>
#include<stdio.h>
#include<vector>
//#include"./ReadSTLfile/Point3f.h"

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <learnopengl/shader_m.h>
//#include <learnopengl/camera.h>
#include"./OpenGL/Mycamera.h"
//#include <learnopengl/model.h>
#include<learnopengl/filesystem.h>

using namespace std;

#define ALIGN(x)	__align__(x)
#define ID_UNDEFI	0xFFFF
#define ID_UNDEFL	0xFFFFFFFF
#define ID_UNDEF64	0xFFFFFFFFFFFFFFFF
#define CHAN_UNDEF	255
#define MAX_CHANNEL  32

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void mouse_callback(GLFWwindow* window, double xpos, double ypos);
void scroll_callback(GLFWwindow* window, double xoffset, double yoffset);
void processInput(GLFWwindow* window);

unsigned int SCR_WIDTH = 1200;
unsigned int SCR_HEIGHT = 900;

// camera
Mycamera camera = Mycamera(glm::vec3(30.0f, 30.0f, 80.0f));
float lastX = SCR_WIDTH / 2.0f;
float lastY = SCR_HEIGHT / 2.0f;
bool firstMouse = true;

// timing
float deltaTime = 0.0f;
float lastFrame = 0.0f;

glm::vec3 lightPos = glm::vec3(12.0f, 10.0f, 20.0f);

struct Point3f {
	float x;
	float y;
	float z;
};

/*
struct Octree {
	Point3f origin;         //! The physical center of this node
	Point3f halfDimension;  //! Half the width/height/depth of this node 体素的长宽高的半值
	Octree* children[8]; //! Pointers to child octants
	int accuracy = 0;
	bool exist = true;//该节点是否存在;
	bool sub = false;//是否已被分割

	Octree(Point3f O, Point3f HD) {
		origin = O;
		halfDimension = HD;
		for (int i = 0; i <= 7; ++i) {
			children[i] = nullptr;
		}

		exist = true;
		sub = false; 
	}

};
*/

struct Octree {
	Point3f origin;         //! The physical center of this node
	Point3f halfDimension;  //! Half the width/height/depth of this node 体素的长宽高的半值
	Octree* children[8]; //! Pointers to child octants
	int accuracy = 0;
	bool exist = true;//该节点是否存在;
	bool sub = false;//是否已被分割
	Octree(Point3f O, Point3f HD) {
		origin = O;
		halfDimension = HD;

		for (int i = 0; i <= 7; ++i) {
			children[i] = nullptr;
		}
		exist = true;
		sub = false;
	}

};

struct BoundBox
{
	Point3f Origin;
	float length;//x
	float width;//y
	float height;//z
};

struct Myvector
{
	int buf_len_;//数组容量大小
	int cnt_top_;//数组已经装填的大小
	//Point3f* buf_;
};


BoundBox GetCutterBoundBox(Point3f& Center, Point3f& CutterSize);
Point3f GetCutterPos(float T, float t, Point3f& InitCutPosition, Point3f& origin, Point3f& halfDimension);

__device__  void Push(Myvector* obj, Point3f target, Point3f* buffer)
{
	if (obj->cnt_top_ + 1 == obj->buf_len_) {

		//增加容量
		Point3f* temp;
		hipMalloc((void**)&(temp), obj->buf_len_ * 2 * sizeof(Point3f));
		obj->buf_len_ = obj->buf_len_ * 2;
		//hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind);
		//hipMemcpy为host函数，不能在此使用
		hipMemcpyAsync(temp, buffer, (obj->cnt_top_ + 1)* sizeof(Point3f), hipMemcpyDeviceToDevice);
		hipFree(buffer);
		buffer = temp;
	}
	buffer[obj->cnt_top_] = target;

	obj->cnt_top_++;

	//printf("%d号装填成功", obj->cnt_top_);
	/*for (size_t i = 0; i < obj.buf_len_; ++i)
	temp[i] = obj.buf_[i];
*/
}

//在GPU中调用，判断刀具包围盒是否与体素相交
__device__ bool IsIntersect(Octree* Voxel, BoundBox& CutterBox) {
	//printf("调用IsIntersect：");
	if (std::abs(Voxel->origin.x - CutterBox.Origin.x) < std::abs(Voxel->halfDimension.x + (CutterBox.length / 2))
		&& std::abs(Voxel->origin.y - CutterBox.Origin.y) < std::abs(Voxel->halfDimension.y + (CutterBox.width / 2))
		&& std::abs(Voxel->origin.z - CutterBox.Origin.z) < std::abs(Voxel->halfDimension.z + (CutterBox.height / 2))
		)
	{
		//printf("相交\n");
		return true;
	}
	else
	{
		//printf("不相交\n");
		return false;
	}

}

//此处可以设置成八个点并行判断，设置同步，等待8个点判断完毕后在返回，输入该节点的指针，根据线程号来计算顶点
__global__  void IsIn(Octree* curr, int i, Point3f& CutPosition, Point3f& CutterSize) {

	Point3f OctreeVer;
	OctreeVer.x = curr->origin.x - curr->halfDimension.x;
	OctreeVer.x = OctreeVer.x + (threadIdx.x & 4) / 2 * curr->halfDimension.x;

	OctreeVer.y = curr->origin.y - curr->halfDimension.y;
	OctreeVer.y = OctreeVer.y + (threadIdx.x & 2) * curr->halfDimension.y;

	OctreeVer.z = curr->origin.z - curr->halfDimension.z;
	OctreeVer.z = OctreeVer.z + (threadIdx.x & 1) * 2 * curr->halfDimension.z;

	//return OctreeVer;

	float X = abs(OctreeVer.x - CutPosition.x);
	float Y = abs(OctreeVer.y - CutPosition.y);
	float Z = (OctreeVer.z - CutPosition.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > CutterSize.y || Y > CutterSize.y || Z > CutterSize.x ||
		(X * X + Y * Y + Z * Z) >= (CutterSize.y * CutterSize.y))
	{
		curr->sub = true;
	}


	//该顶点在刀具内
	//curr->sub一旦为true，那么该节点必须被分割，经过8个节点的判断后仍然为false，说明该体素八个顶点都在刀具内，不需要被分割，需要被移除
	//curr->sub = false;
	//块内同步函数，同一block内所有线程执行至__syncthreads()处等待全部线程执行完毕后再继续
	//__syncthreads();
}

//串行处理8个点，输入当前要判断的顶点，输出它是否与刀具相交
__device__  bool IsIn_a(Octree* curr, int i, Point3f& CutPosition, Point3f& CutterSize) {

	Point3f OctreeVer;
	OctreeVer.x = curr->origin.x - curr->halfDimension.x;
	OctreeVer.x = OctreeVer.x + (i & 4) / 2 * curr->halfDimension.x;

	OctreeVer.y = curr->origin.y - curr->halfDimension.y;
	OctreeVer.y = OctreeVer.y + (i & 2) * curr->halfDimension.y;

	OctreeVer.z = curr->origin.z - curr->halfDimension.z;
	OctreeVer.z = OctreeVer.z + (i & 1) * 2 * curr->halfDimension.z;

	float X = abs(OctreeVer.x - CutPosition.x);
	float Y = abs(OctreeVer.y - CutPosition.y);
	float Z = (OctreeVer.z - CutPosition.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > CutterSize.y || Y > CutterSize.y || Z > CutterSize.x ||
		(X * X + Y * Y + Z * Z) >= (CutterSize.y * CutterSize.y))
	{
		curr->sub = true;
		return false;
	}
	return true;

	//该顶点在刀具内
	//curr->sub一旦为true，那么该节点必须被分割，经过8个节点的判断后仍然为false，说明该体素八个顶点都在刀具内，不需要被分割，需要被移除
	//curr->sub = false;
	//块内同步函数，同一block内所有线程执行至__syncthreads()处等待全部线程执行完毕后再继续
	//__syncthreads();
}

__device__  void InitParent(int index, Octree* GrandNode) {

	printf("分配内存初始化\n");
	hipMalloc((void**)&(GrandNode->children[index]), sizeof(Octree));
	//hipMalloc((void**)&(GrandNode->children[index]->children), 64 * sizeof(Octree));
	GrandNode->children[index]->accuracy = GrandNode->accuracy + 1;

	//printf("allocMemeoty,accuracy:%d\n", GrandNode->children[threadIdx.x]->accuracy);
	//计算子节点的原点
	GrandNode->children[index]->origin.x = GrandNode->origin.x - GrandNode->halfDimension.x * 3 / 4;
	GrandNode->children[index]->origin.x = GrandNode->children[index]->origin.x + ((threadIdx.x & 3) * GrandNode->halfDimension.x) / 2;

	GrandNode->children[index]->origin.y = GrandNode->origin.y - GrandNode->halfDimension.y * 3 / 4;
	GrandNode->children[index]->origin.y = GrandNode->children[index]->origin.y + ((threadIdx.x & 12) / 4 * GrandNode->halfDimension.y) / 2;

	GrandNode->children[index]->origin.z = GrandNode->origin.z - GrandNode->halfDimension.z * 3 / 4;
	GrandNode->children[index]->origin.z = GrandNode->children[index]->origin.z + ((threadIdx.x & 48) / 16 * GrandNode->halfDimension.z) / 2;

	//计算尺寸
	GrandNode->children[index]->halfDimension.x = GrandNode->halfDimension.x / 4;
	GrandNode->children[index]->halfDimension.y = GrandNode->halfDimension.y / 4;
	GrandNode->children[index]->halfDimension.z = GrandNode->halfDimension.z / 4;

	//设定初始参数
	GrandNode->children[index]->exist = true;
	GrandNode->children[index]->sub = false;

	for (int i = 0; i <= 63; ++i) {
		GrandNode->children[index]->children[i] = nullptr;
	}
}
//对节点进行判断，若需要分割则会创建8个线程来分别对8个子节点进行处理，通过线程号进行区别
__global__ void Cutter_Dynamic(Octree* GrandNode, BoundBox& CutterBox, Point3f& CutPosition, Point3f& CutterSize) {

	//if (GrandNode->accuracy > 4) return;
	//为被分割出来的一个子节点分配显存。
	//if(!alloc)
	if ((GrandNode->children[threadIdx.x]) == nullptr)
	{
		hipMalloc((void**)&(GrandNode->children[threadIdx.x]), sizeof(Octree));
		GrandNode->children[threadIdx.x]->accuracy = GrandNode->accuracy + 1;
		//printf("allocMemeoty,accuracy:%d\n", GrandNode->children[threadIdx.x]->accuracy);
		//计算子节点的原点
		GrandNode->children[threadIdx.x]->origin.x = GrandNode->origin.x - GrandNode->halfDimension.x / 2;
		GrandNode->children[threadIdx.x]->origin.x = GrandNode->children[threadIdx.x]->origin.x + (threadIdx.x & 1) * GrandNode->halfDimension.x;
		GrandNode->children[threadIdx.x]->origin.y = GrandNode->origin.y - GrandNode->halfDimension.y / 2;
		GrandNode->children[threadIdx.x]->origin.y = GrandNode->children[threadIdx.x]->origin.y + (threadIdx.x & 2) * GrandNode->halfDimension.y / 2;
		GrandNode->children[threadIdx.x]->origin.z = GrandNode->origin.z - GrandNode->halfDimension.z / 2;
		GrandNode->children[threadIdx.x]->origin.z = GrandNode->children[threadIdx.x]->origin.z + (threadIdx.x & 4) * GrandNode->halfDimension.z / 4;

		//计算尺寸
		GrandNode->children[threadIdx.x]->halfDimension.x = GrandNode->halfDimension.x / 2;
		GrandNode->children[threadIdx.x]->halfDimension.y = GrandNode->halfDimension.y / 2;
		GrandNode->children[threadIdx.x]->halfDimension.z = GrandNode->halfDimension.z / 2;

		//设定初始参数
		GrandNode->children[threadIdx.x]->exist = true;
		GrandNode->children[threadIdx.x]->sub = false;

		GrandNode->children[threadIdx.x]->children[0] = nullptr;
		GrandNode->children[threadIdx.x]->children[1] = nullptr;
		GrandNode->children[threadIdx.x]->children[2] = nullptr;
		GrandNode->children[threadIdx.x]->children[3] = nullptr;
		GrandNode->children[threadIdx.x]->children[4] = nullptr;
		GrandNode->children[threadIdx.x]->children[5] = nullptr;
		GrandNode->children[threadIdx.x]->children[6] = nullptr;
		GrandNode->children[threadIdx.x]->children[7] = nullptr;
		
	}
	//子节点的精细度+1
	//*printf("精度为%d", GrandNode->children[threadIdx.x]->accuracy);
	//GrandNode->children[threadIdx.x]->children = nullptr;
	if (GrandNode->children[threadIdx.x]->accuracy >= 7 || !(GrandNode->children[threadIdx.x]->exist)) return;

	if (IsIntersect(GrandNode->children[threadIdx.x], CutterBox)) {
		//printf("判断完毕\n");
		for (int i = 0; i <= 7; i++) {

			if (!IsIn_a(GrandNode->children[threadIdx.x], i, CutPosition, CutterSize) )  {
				
				//分割出8个子节点的子节点
				//*printf("分割\n");
				bool alloc = GrandNode->children[threadIdx.x]->sub;//判断其是否已经被分内存
				GrandNode->children[threadIdx.x]->sub = true;
				//printf("分割:精度%d", GrandNode->children[threadIdx.x]->accuracy);
				Cutter_Dynamic << < 1, 8 >> > (GrandNode->children[threadIdx.x], CutterBox, CutPosition, CutterSize);
				return;

			}
			else {
				if (i == 7) {
					//8个点都在内部
					GrandNode->children[threadIdx.x]->exist = false;
					return;
				}
				continue;
			}
		}

	}
	return;
}

__global__ void Cutter_Dynamic_64(Octree* GrandNode, BoundBox& CutterBox, Point3f& CutPosition, Point3f& CutterSize,bool alloc) {

	//为被分割出来的一个子节点分配显存。
	//printf("开始分割：\n");
	//int index = blockIdx.x * 16 + (threadIdx.x) * 4 + (threadIdx.y);
	int index = threadIdx.x;
	//if(!alloc)
    if ((GrandNode->children[index]) == nullptr ){
		//printf("分配内存初始化\n");
		hipMalloc((void**)&(GrandNode->children[index]), sizeof(Octree));
		GrandNode->children[index]->accuracy = GrandNode->accuracy + 1;
		//printf("allocMemeoty,accuracy:%d\n", GrandNode->children[threadIdx.x]->accuracy);
		//计算子节点的原点
		GrandNode->children[index]->origin.x = GrandNode->origin.x - GrandNode->halfDimension.x * 3 / 4;
		GrandNode->children[index]->origin.x = GrandNode->children[index]->origin.x + ((threadIdx.x & 3) * GrandNode->halfDimension.x) /2;

		GrandNode->children[index]->origin.y = GrandNode->origin.y - GrandNode->halfDimension.y * 3 / 4;
		GrandNode->children[index]->origin.y = GrandNode->children[index]->origin.y + ((threadIdx.x & 12) / 4 * GrandNode->halfDimension.y) /2;

		GrandNode->children[index]->origin.z = GrandNode->origin.z - GrandNode->halfDimension.z * 3 / 4;
		GrandNode->children[index]->origin.z = GrandNode->children[index]->origin.z + ((threadIdx.x & 48) / 16 * GrandNode->halfDimension.z)/2 ;

		//计算尺寸
		GrandNode->children[index]->halfDimension.x = GrandNode->halfDimension.x / 4;
		GrandNode->children[index]->halfDimension.y = GrandNode->halfDimension.y / 4;
		GrandNode->children[index]->halfDimension.z = GrandNode->halfDimension.z / 4;

		//设定初始参数
		GrandNode->children[index]->exist = true;
		GrandNode->children[index]->sub = false;
		for (int i = 0; i <= 63; ++i) {
			GrandNode->children[index]->children[i] = nullptr;
		}
	}
	//子节点的精细度+1
	//printf("精度为%d\n", GrandNode->children[threadIdx.x]->accuracy);
	if (GrandNode->children[index]->accuracy >= 3 || !GrandNode->children[index]->exist) return;

	if (IsIntersect(GrandNode->children[index], CutterBox)) {
		//printf("判断完毕\n");
		for (int i = 0; i <= 7; i++) {
			if (!IsIn_a(GrandNode->children[index], i, CutPosition, CutterSize)) {
				
				//*printf("分割\n");
				bool alloc = GrandNode->children[index]->sub;//判断其是否已经被分内存
				GrandNode->children[index]->sub = true;
				//printf("分割:精度%d\n", GrandNode->children[threadIdx.x]->accuracy);
				dim3 grid(1);
				dim3 block(64);
				Cutter_Dynamic_64 << < grid,block >> > (GrandNode->children[index], CutterBox, CutPosition, CutterSize, alloc);
				return;
			}
			else {
				if (i == 7) {
					//8个点都在内部
					//printf("*****  剔除体素 *****\n");
					GrandNode->children[index]->exist = false;
					return;
				}
				continue;
			}
		}

	}
	return;
}

__device__ 	void GetOctreeVertex(int number, Octree* curr, Point3f& OctreeVertex) {
	OctreeVertex.x = curr->origin.x - curr->halfDimension.x;
	OctreeVertex.x = OctreeVertex.x + (number & 4) / 2 * curr->halfDimension.x;

	OctreeVertex.y = curr->origin.y - curr->halfDimension.y;
	OctreeVertex.y = OctreeVertex.y + (number & 2) * curr->halfDimension.y;

	OctreeVertex.z = curr->origin.z - curr->halfDimension.z;
	OctreeVertex.z = OctreeVertex.z + (number & 1) * 2 * curr->halfDimension.z;

}

__device__ void IteraAddPoints(Octree* curr, Myvector* testVertex, Point3f* buffer) {
	//printf("添加点:\n");
	if (!curr->sub) {
		Point3f OctreeVertex;
		GetOctreeVertex(0, curr, OctreeVertex);

		Point3f halfDimension = curr->halfDimension;
		Push(testVertex, OctreeVertex, buffer); Push(testVertex, curr->halfDimension, buffer);
		GetOctreeVertex(7, curr, OctreeVertex);

		halfDimension.x *= (-1); halfDimension.y *= (-1); halfDimension.z *= (-1);
		Push(testVertex, OctreeVertex, buffer);Push(testVertex, halfDimension, buffer);
	}
	else
	{
		for (int i = 0; i <= 7; ++i) {
			if (curr->children[i]->exist){
				//printf("探索下一个子节点：当前深度%d\n", curr->children[i]->accuracy);
				IteraAddPoints(curr->children[i], testVertex, buffer);
			}
		}
	}

	//printf("传入函数前(%f,", test.x);
//printf("%f,", test.y);
//printf("%f)\n", test.z);
//Push(testVertex, test);
//printf("压入四个点\n");
}

__global__ void AddPoints(Octree* curr, Myvector* testVertex,Point3f* buffer) {

	//printf("开始遍历：\n");
	//testVertex->buf_len_ = 2000000;
	//testVertex->cnt_top_ = 0;
	
	IteraAddPoints(curr, testVertex, buffer);
}
__global__ void TestVector(Myvector* testVertex, Point3f* buffer) {

	//buffer[0].x = 55;
	//buffer[0].y = 55;
	//buffer[0].z = 55;
	for (int i = 0; i < testVertex->cnt_top_; ++i) {
		//testVertex->buf_[i];

		printf("buffer[%d]: ",i);
		printf("(%f,", buffer[i].x);
		printf("%f,",  buffer[i].y);
		printf("%f)\n", buffer[i].z);
	}
}


__global__ void InitRoot(Octree* Root, BoundBox& CutterBox, Point3f& CutterPos, Point3f& CutterSize) {

	bool alloc;
	printf("Root大小：%d",sizeof(Octree));
	printf("RootSub:%d\n", Root->sub);
	if (!Root->sub)
	{
		alloc = false;
		Root->sub = true;
	}
	Root->accuracy = 0;
	Root->exist = true;
	dim3 grid(1);
	dim3 block(64);
	Cutter_Dynamic << < 1, 8 >> > (Root, CutterBox, CutterPos, CutterSize);
	//Cutter_Dynamic_64 <<< grid, block >> > (Root, CutterBox, CutterPos, CutterSize, Root->sub);

}


//在CPU计算得到包围盒，放在共享内存中
BoundBox GetCutterBoundBox(Point3f& Center, Point3f& CutterSize)//暂时不考虑刀具的旋转,输入刀具的位置和大小参数
{
	//暂时不考虑刀具的旋转；
	BoundBox box;

	box.Origin.x = Center.x;
	box.Origin.y = Center.y;
	box.Origin.z = Center.z + (CutterSize.x - CutterSize.y) / 2;

	box.length = 2 * CutterSize.y;
	box.width = 2 * CutterSize.y;
	box.height = CutterSize.y + CutterSize.x;

	return box;
}
Point3f GetCutterPos(float T, float t, Point3f& InitCutPosition, Point3f& origin, Point3f& halfDimension) {

	float part = t / T;

	Point3f res;
	res.x = InitCutPosition.x + part * halfDimension.x * 2 * 3 / 2;
	res.y = InitCutPosition.y + part * halfDimension.y * 2 * 3 / 2;
	res.z = InitCutPosition.z;

	/*return Point3f(InitCutPosition.x + part * halfDimension.x * 2 * 3 / 2,
		InitCutPosition.y + part * halfDimension.y * 2 * 3 / 2,
		InitCutPosition.z);//单位为mm;
	*/
	return res;
}

//使用刀具分割切削八叉树，获得切削后的八叉树节点

int Run_GPU_Octree() {

	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 4);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);


#ifdef __APPLE__
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return 0;
	}
	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	glfwSetCursorPosCallback(window, mouse_callback);
	glfwSetScrollCallback(window, scroll_callback);

	// tell GLFW to capture our mouse
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

	// glad: load all OpenGL function pointers
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
			std::cout << "Failed to initialize GLAD" << std::endl;
			return 0;
	}

	// configure global opengl state
	// -----------------------------
	glEnable(GL_DEPTH_TEST);

	camera.Right += 100;//调整相机视角

	GLuint VBO[3], VAO[3], EBO;

	glGenVertexArrays(3, VAO);
	glGenBuffers(3, VBO);
	glGenBuffers(1, &EBO);

	Shader ourShader("./res/Octree_DrawPoints.vs", "./res/Octree_DrawPoints.fs", "./res/Octree_DrawPoints.gs");
	


	Point3f CutPosition, HostCutterSize, HostCutterPos;
	BoundBox HostCutterBox;

	Point3f origin, halfDimension;
	//设置工件初始位置
	origin.x = 30.0f; origin.y = 30.0f; origin.z = 30.0f;
	//设置工件尺寸
	halfDimension.x = 20.0f; halfDimension.y = 20.0f; halfDimension.z = 20.0f;
	Octree temp(origin, halfDimension);
	temp.sub = true;//重要
	Octree* HostRoot = &temp;

	CutPosition.x = 20;
	CutPosition.y = 20;
	CutPosition.z = 50;

	HostCutterSize.x = 8;
	HostCutterSize.y = 6;
	HostCutterSize.z = 6;


  

	//开始切割，刀具运动
	
	float  T = 1000;//设为1000
	float delta_t = 2;//步长设为2
	float t = 0;
	HostCutterPos = GetCutterPos(T, t, CutPosition,origin,halfDimension);
	
	Octree* DeviceRoot;
	Point3f* DeviceCutterSize;
	Point3f* DeviceCutterPos;
	BoundBox* DeviceCutterBox;
	

	hipMalloc((void**)&(DeviceRoot), sizeof(Octree));
	hipMemcpy(DeviceRoot, HostRoot, sizeof(Octree), hipMemcpyHostToDevice);
	bool RootSub = false;
	hipMemcpy(&(DeviceRoot->sub), &RootSub, sizeof(bool), hipMemcpyHostToDevice);
	hipMalloc((void**)&(DeviceCutterSize), sizeof(Point3f));
	hipMalloc((void**)&(DeviceCutterPos), sizeof(Point3f));
	hipMalloc((void**)&(DeviceCutterBox), sizeof(BoundBox));
	
	
	Myvector* HostVector;
	Myvector* DeviceVector;
	hipHostMalloc((void**)&HostVector, sizeof(Myvector));
	hipMalloc((void**)&(DeviceVector), sizeof(Myvector));

	const int BufferLen = 40000;

	HostVector->buf_len_ = BufferLen;
	HostVector->cnt_top_ = 0;

	Point3f* D_buffer;//Device端数组
	hipMalloc((void**)&(D_buffer), BufferLen * sizeof(Point3f));

	Point3f* testvector;//host端的数组
	testvector = (Point3f*)malloc(BufferLen * sizeof(Point3f));


	//for (t = 0 ; t <= T;t+= delta_t) 

	while (!glfwWindowShouldClose(window))
	{
		
		t += delta_t;
		cout << "循环" << t / 2 << "次\n" << endl;
		HostCutterPos = GetCutterPos(T, t, CutPosition, origin, halfDimension);
		//t += delta_t;
		HostCutterBox = GetCutterBoundBox(HostCutterPos, HostCutterSize);
		//cout << "调用Cutter_Dynamic" << endl;
		glm::mat4 projection = glm::perspective(glm::radians(camera.Zoom), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 100.0f);
		glm::mat4 view = camera.GetViewMatrix();

		ourShader.use();
		ourShader.setMat4("projection", projection);
		ourShader.setMat4("view", view);
		ourShader.setMat4("projectionInverse", glm::inverse(projection));
		ourShader.setMat4("viewInverse", glm::inverse(view));
		glm::mat4 model = glm::mat4(1.0f);

		model = glm::translate(model, lightPos); // translate it down so it's at the center of the scene
		model = glm::scale(model, glm::vec3(0.5f, 0.5f, 0.5f));	// it's a bit too big for our scene, so scale it down
		ourShader.setMat4("model", model);

		//设置光照参数
		//ourShader.setVec3("lightPos", glm::vec3(60.5f, 60.5f, 30.5f));
		ourShader.setVec3("lightPos", glm::vec3(-80.0f, -80.0f, -80.0f));
		ourShader.setVec3("lightColor", glm::vec3(1.0f, 1.0f, 1.0f));
		ourShader.setVec3("objectColor", glm::vec3(1.0f, 0.5f, 0.31f));

		processInput(window);

		// render
		// ------
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

		glUseProgram(ourShader.ID);


		hipMemcpy(DeviceCutterBox, &HostCutterBox, sizeof(BoundBox), hipMemcpyHostToDevice);
		hipMemcpy(DeviceCutterSize, &HostCutterSize, sizeof(Point3f), hipMemcpyHostToDevice);
		hipMemcpy(DeviceCutterPos, &HostCutterPos, sizeof(Point3f), hipMemcpyHostToDevice);
		
		//可以考虑将存储可渲染点的数组分成8个，最后再进行合并
		//Cutter_Dynamic << < 1,8 >> > (DeviceRoot, *DeviceCutterBox, *DeviceCutterPos, *DeviceCutterSize);
		/*dim3 grid(1);
		dim3 block(64);
		Cutter_Dynamic_64 << < grid, block >> > (DeviceRoot, *DeviceCutterBox, *DeviceCutterPos, *DeviceCutterSize);
		*/
		InitRoot << < 1,1 >> > (DeviceRoot, *DeviceCutterBox, *DeviceCutterPos, *DeviceCutterSize);
		hipDeviceSynchronize();//该方法将停止CPU端线程的执行，直到GPU端完成之前CUDA的任务，包括kernel函数、数据拷贝等

		//此时可以获得分割后的八叉树，之后需要遍历八叉树获得可以被渲染的体素
	
	    //设置数组初始尺寸
	    //创建数组，分配内存
		hipMemcpy(DeviceVector, HostVector, sizeof(Myvector), hipMemcpyHostToDevice);

		AddPoints <<<1, 1 >>> (DeviceRoot, DeviceVector, D_buffer);
		hipDeviceSynchronize();
		//TestVector<<<1,1>>>(DeviceVector);
		
		//Point3f* Mypoints =nullptr;
		//hipMemcpy(&HostVector, DeviceVector, sizeof(Myvector), hipMemcpyDeviceToHost);//复制device端的参数

		int PointNum = 0;
		//hipMemcpy(HostVector, DeviceVector,  sizeof(Myvector), hipMemcpyDeviceToHost);
		//PointNum = (HostVector->cnt_top_)[0];
		hipMemcpy(&PointNum, &(DeviceVector->cnt_top_), sizeof(int), hipMemcpyDeviceToHost);
		cout << "D_buffer:" << endl;
		cout << "获得 " << PointNum << " 个点" << endl;
		
		//TestVector << <1, 1 >> > (DeviceVector, D_buffer);
		//hipDeviceSynchronize();
		//hipHostMalloc((void**)&testvector, PointNum * sizeof(Point3f));
		//testvector = (Point3f*)malloc(HostVector.cnt_top_ * sizeof(Point3f));

		hipMemcpy(testvector, D_buffer, PointNum * sizeof(Point3f), hipMemcpyDeviceToHost);
		cout << "";
	
		//hipMemcpy用于在主机（Host）和设备（Device）之间往返的传递数据，用法如下：
		//主机到设备：hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice)
		//设备到主机：hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost)
		//注意：该函数是同步执行函数，在未完成数据的转移操作之前会锁死并一直占有CPU进程的控制权，所以不用再添加hipDeviceSynchronize()函数

		//if (HostVector->cnt_top_ != 0){
	    //HostVector.buf_ = (Point3f*)malloc((HostVector.cnt_top_) * sizeof(Point3f));
		
				   //顶点绘制
		glBindVertexArray(VAO[2]);//绑定第一个顶点对象Id到顶点对象，顶点对象包含了顶点属性设置，缓冲对象设置等一系列属性
		glBindBuffer(GL_ARRAY_BUFFER, VBO[2]);//绑定缓冲对象
		//设置缓冲区需要存储的顶点集，用于后续传送给GPU
		glBufferData(GL_ARRAY_BUFFER, PointNum * sizeof(Point3f), testvector, GL_STATIC_DRAW);
		//设置顶点位置属性，指定location为0
		glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 2 * sizeof(Point3f), (void*)0);//体素的一个顶点
		glEnableVertexAttribArray(0);
		glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 2 * sizeof(Point3f), (void*)(sizeof(Point3f)));//体素的长宽高
		glEnableVertexAttribArray(1);
		glBindBuffer(GL_ARRAY_BUFFER, 0);

		glBindVertexArray(0);
		glBindVertexArray(VAO[2]);
		glDrawArrays(GL_POINTS, 0, PointNum);

		//cout << "获得 " << DeviceVector->cnt_top_ << " 个点" << endl;
		glfwSwapBuffers(window);
		glfwPollEvents();

		hipMemcpy(DeviceVector, HostVector, sizeof(Myvector), hipMemcpyHostToDevice);
		//free(testvector);
		//hipFree(D_buffer);
	    //}
	}

	glfwTerminate();

	return 0;
	

	/*for (int i = 0; testVector.cnt_top_ > i; i += 2) {
	cout << "点[" << i << "]位置 (" << (&Mypoints)[i]->x << "," << (&Mypoints)[i]->y << "," << (&Mypoints)[i]->z << ")"
		<< "点[" << i << "]尺寸 (" << (&Mypoints)[i + 1]->x << "," << (&Mypoints)[i + 1]->y << "," << (&Mypoints)[i + 1]->z << ")\n";
}*/

	return 0;

}

int main() {
	hipEvent_t start, stop;
	float elapseTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	Run_GPU_Octree();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapseTime, start, stop);
	system("pause");
	return 0;
}



void  processInput(GLFWwindow* window)
{
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
		glfwSetWindowShouldClose(window, true);

	if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
		camera.ProcessKeyboard(FORWARD, deltaTime);
	if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
		camera.ProcessKeyboard(BACKWARD, deltaTime);
	if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
		camera.ProcessKeyboard(LEFT, deltaTime);
	if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
		camera.ProcessKeyboard(RIGHT, deltaTime);
	if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS)
		camera.ProcessKeyboard(UP, deltaTime);
	if (glfwGetKey(window, GLFW_KEY_E) == GLFW_PRESS)
		camera.ProcessKeyboard(DOWN, deltaTime);

}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void  framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	// make sure the viewport matches the new window dimensions; note that width and 
	// height will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, height);
}

// glfw: whenever the mouse moves, this callback is called
// -------------------------------------------------------
void mouse_callback(GLFWwindow* window, double xpos, double ypos)
{
	if (firstMouse)
	{
		lastX = xpos;
		lastY = ypos;
		firstMouse = false;
	}

	float xoffset = xpos - lastX;
	float yoffset = lastY - ypos; // reversed since y-coordinates go from bottom to top

	lastX = xpos;
	lastY = ypos;

	camera.ProcessMouseMovement(xoffset, yoffset);
}

// glfw: whenever the mouse scroll wheel scrolls, this callback is called
// ----------------------------------------------------------------------
void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{
	camera.ProcessMouseScroll(yoffset);
}



//printf("ֵ:%d", ret[threadIdx.x]);
//printf("from AplusThree function\n");
//printf(" Octree* test：%d  \n" ,test->accuracy);
//printf("threadIdx.x: %d \n", threadIdx.x);

/*//初始化节点
void InitRootNode(Point3f& CutPosition, Point3f& CutterSize, Octree* Root) {

	Root->origin.x = 30.0f;  //设置初始位置
	Root->origin.y = 30.0f;
	Root->origin.z = 30.0f;

	Root->halfDimension.x = 20.0f;
	Root->halfDimension.y = 20.0f;
	Root->halfDimension.z = 20.0f;

	Root->accuracy = 0;
	Root->sub = true;

	CutPosition = Point3f(20, 20, 50);
	CutterSize = Point3f(8, 6, 6); 

}

//T = 1000; delta_t = 2
void RunCutter(float TT,float Delta_t, Point3f& CutPosition, Point3f& CutterSize, Octree* Root) {

	float  T = TT;//设为1000
	float delta_t = Delta_t;//设为2
	float t = 0;
	Point3f CutterPos = GetCutterPos(T, t,CutPosition,Root);
	while (true) {
		CutterPos = GetCutterPos(T, t, CutPosition, Root);
		t += delta_t;
		BoundBox CutterBox = GetCutterBoundBox(CutterPos, CutterSize);
		Cutter_Dynamic << < 1, 8 >> > (Root, CutterBox, CutterPos, CutterSize);//可以考虑将存储可渲染点的数组分成8个，最后再进行合并

	}
	
}
*/