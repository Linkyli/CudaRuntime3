#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include"hip/device_functions.h"
#include <iostream>
#include <Windows.h>
#include<stdio.h>
#include<vector>
#include"./ReadSTLfile/Point3f.h"
//#include"./Octree/Octree.h"

using namespace std;

#define ALIGN(x)	__align__(x)
#define ID_UNDEFI	0xFFFF
#define ID_UNDEFL	0xFFFFFFFF
#define ID_UNDEF64	0xFFFFFFFFFFFFFFFF
#define CHAN_UNDEF	255
#define MAX_CHANNEL  32

/*struct ALIGN(16) VDBNode {
    char		mLev;			// Level		Max = 255			1 byte
	char		mFlags;
	char		mPriority;
	char		pad;
	int3		mPos;			// Pos			Max = +/- 4 mil (linear space/range)	12 bytes
	int3		mValue;			// Value		Max = +8 mil		4 bytes
	float3		mVRange;
	__int64 mParent;		// Parent ID						8 bytes
	__int64		mChildList;		// Child List						8 bytes
	__int64		mMask;			// Bitmask starts - Must keep here, even if not USE_BITMASKS
};*/
/*__global__ void GetRenderedCube(Octree* test, )
{

}*/

struct Octree {
	float3 origin;         //! The physical center of this node
	float3 halfDimension;  //! Half the width/height/depth of this node 体素的长宽高的半值
	Octree* children[8]; //! Pointers to child octants
	int accuracy = 0;
	bool exist = true;//该节点是否已被剔除
	bool sub = false;//是否已被分割
};


struct BoundBox
{
	float3 Origin;
	float length;//x
	float width;//y
	float height;//z
};
/*Point3f CutterSize;//放在共享内存中
Point3f CutPosition;
BoundBox CutterBox;
*/
//在CPU计算得到包围盒，放在共享内存中
BoundBox GetCutterBoundBox(float3& Center, float3& CutterSize)//暂时不考虑刀具的旋转,输入刀具的位置和大小参数
{
	//暂时不考虑刀具的旋转；
	BoundBox box;
	box.Origin = Center;
	box.Origin.z = Center.z + (CutterSize.x - CutterSize.y) / 2;

	box.length = 2 * CutterSize.y;
	box.width = 2 * CutterSize.y;
	box.height = CutterSize.y + CutterSize.x;

	return box;
}
struct Myvector
{
	size_t buf_len_;//数组容量大小
	size_t cnt_top_;//数组已经装填的大小
	float3* buf_;
};
__device__  void Push(Myvector& obj, float3 target)
{
	printf("运行\n");

	if (obj.cnt_top_ + 1 == obj.buf_len_) {

		printf("增加容量\n");
		float3* temp;
		hipMalloc((void**)&(temp), obj.buf_len_ * 1.5);
		//T* tmp = new T[buf_len_ * 1.5];
		obj.buf_len_ = obj.buf_len_ * 1.5;
		for (size_t i = 0; i < obj.buf_len_; ++i)
			temp[i] = obj.buf_[i];

		free(obj.buf_);
		obj.buf_ = temp;
	}

	obj.buf_[++obj.cnt_top_] = target;
	printf("%d号装填成功", obj.cnt_top_);
}

//在GPU中调用，判断刀具包围盒是否与体素相交
__device__ bool IsIntersect(Octree* Voxel, BoundBox& CutterBox){
	if (std::abs(Voxel->origin.x - CutterBox.Origin.x) < std::abs(Voxel->halfDimension.x + (CutterBox.length / 2))
		&& std::abs(Voxel->origin.y - CutterBox.Origin.y) < std::abs(Voxel->halfDimension.y + (CutterBox.width / 2))
		&& std::abs(Voxel->origin.z - CutterBox.Origin.z) < std::abs(Voxel->halfDimension.z + (CutterBox.height / 2))
		)
	{
		//cout << "相交" << endl;
		return true;
	}
	else
	{
		//cout << "不相交" << endl;
		return false;
	}
}

//此处可以设置成八个点并行判断，设置同步，等待8个点判断完毕后在返回，输入该节点的指针，根据线程号来计算顶点
__global__  void IsIn(Octree* curr, int i, Point3f& CutPosition, Point3f& CutterSize) {

	float3 OctreeVer;
	OctreeVer.x = curr->origin.x - curr->halfDimension.x;
	OctreeVer.x = OctreeVer.x + (threadIdx.x & 4) / 2 * curr->halfDimension.x;

	OctreeVer.y = curr->origin.y - curr->halfDimension.y;
	OctreeVer.y = OctreeVer.y + (threadIdx.x & 2) * curr->halfDimension.y;

	OctreeVer.z = curr->origin.z - curr->halfDimension.z;
	OctreeVer.z = OctreeVer.z + (threadIdx.x & 1) * 2 * curr->halfDimension.z;

	//return OctreeVer;

	float X = abs(OctreeVer.x - CutPosition.x);
	float Y = abs(OctreeVer.y - CutPosition.y);
	float Z = (OctreeVer.z - CutPosition.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > CutterSize.y || Y > CutterSize.y || Z > CutterSize.x || 
		(X * X + Y * Y + Z * Z) >= (CutterSize.y * CutterSize.y)) 
	{  curr->sub = true; }


	//该顶点在刀具内
	//curr->sub一旦为true，那么该节点必须被分割，经过8个节点的判断后仍然为false，说明该体素八个顶点都在刀具内，不需要被分割，需要被移除
	//curr->sub = false;
	//块内同步函数，同一block内所有线程执行至__syncthreads()处等待全部线程执行完毕后再继续
	//__syncthreads();
}



//串行处理8个点，输入当前要判断的顶点，输出它是否与刀具相交
__device__  bool IsIn_a(Octree* curr,int i, Point3f &CutPosition, Point3f& CutterSize) {
	
	float3 OctreeVer;
	OctreeVer.x = curr->origin.x - curr->halfDimension.x;
	OctreeVer.x = OctreeVer.x + (i & 4) / 2 * curr->halfDimension.x;

	OctreeVer.y = curr->origin.y - curr->halfDimension.y;
	OctreeVer.y = OctreeVer.y + (i & 2) * curr->halfDimension.y;

	OctreeVer.z = curr->origin.z - curr->halfDimension.z;
	OctreeVer.z = OctreeVer.z + (i & 1) * 2 * curr->halfDimension.z;

	float X = abs(OctreeVer.x - CutPosition.x);
	float Y = abs(OctreeVer.y - CutPosition.y);
	float Z = (OctreeVer.z - CutPosition.z);

	//该顶点不在刀具内->需要被分割->curr->sub = true
	if (X > CutterSize.y || Y > CutterSize.y || Z > CutterSize.x || 
		(X * X + Y * Y + Z * Z) >= (CutterSize.y * CutterSize.y)) 
	{   curr->sub = true;
		return false;
	}
	return true;

	//该顶点在刀具内
	//curr->sub一旦为true，那么该节点必须被分割，经过8个节点的判断后仍然为false，说明该体素八个顶点都在刀具内，不需要被分割，需要被移除
	//curr->sub = false;
	//块内同步函数，同一block内所有线程执行至__syncthreads()处等待全部线程执行完毕后再继续
	//__syncthreads();
}

//对节点进行判断，若需要分割则会创建8个线程来分别对8个子节点进行处理，通过线程号进行区别
__global__ void Cutter_Dynamic(Octree* GrandNode, BoundBox& CutterBox, Point3f& CutPosition, Point3f& CutterSize) {

	//为被分割出来的一个子节点分配显存。
	hipMalloc((void**)&(GrandNode->children[threadIdx.x]), sizeof(Octree));
	//子节点的精细度+1
	GrandNode->children[threadIdx.x]->accuracy = GrandNode->accuracy + 1;

	if (IsIntersect(GrandNode->children[threadIdx.x], CutterBox)) {
		for (int i = 0; i <= 7; i++) {
			if (!IsIn_a(GrandNode->children[threadIdx.x],i, CutPosition, CutterSize)) {
				//有一个节点在外部 -> 相交 ->分割，或许存在误判，后期再改进
				if (GrandNode->children[threadIdx.x]->accuracy <= 3) {

					//分割出8个子节点的子节点
					Cutter_Dynamic <<< 1, 8 >>> (GrandNode->children[threadIdx.x],CutterBox, CutPosition,CutterSize);
					return;
				}
			}
		}
		//8个点都在内部
		GrandNode->children[threadIdx.x]->exist = false;
	}

}


//判断8个顶点是否在刀具内
//IsIn << <1, 8 >> > (GrandNode->children[threadIdx.x]);
//printf(" children[%d]->accuracy  = %d\n", threadIdx.x, test->children[threadIdx.x]->accuracy);

__global__ void TestOctree(int* ret, Octree* test)
{
	//ret[threadIdx.x] += 3;

	hipMalloc((void**)&(test->children[threadIdx.x]) , sizeof(Octree));

	test->children[threadIdx.x]->accuracy = test->accuracy + 1;
	//test->children[threadIdx.x]->sub = true;

	//printf(" children[%d]->accuracy  = %d\n", threadIdx.x, test->children[threadIdx.x]->accuracy);

	if (test->children[threadIdx.x]->accuracy <= 1) {
		TestOctree <<< 1, 8 >> > (ret, test->children[threadIdx.x]);
	}

}

//printf("ֵ:%d", ret[threadIdx.x]);
//printf("from AplusThree function\n");
//printf(" Octree* test：%d  \n" ,test->accuracy);
//printf("threadIdx.x: %d \n", threadIdx.x);

__global__ void AplusB(int* ret, int a, int b, Octree* testB)
{
	ret[threadIdx.x] += a + b + threadIdx.x;
	Octree* test = testB;
	test->exist = false;
	test->accuracy = 1;

	TestOctree <<< 1, 8>>> (ret, test);
	//printf("ֵ:%d", ret[threadIdx.x]);
}
__device__ 	float3 GetOctreeVertex(int number, Octree* curr) {
	float3 OctreeVertex;
	OctreeVertex.x = curr->origin.x - curr->halfDimension.x;
	OctreeVertex.x = OctreeVertex.x + (number & 4) / 2 * curr->halfDimension.x;

	OctreeVertex.y = curr->origin.y - curr->halfDimension.y;
	OctreeVertex.y = OctreeVertex.y + (number & 2) * curr->halfDimension.y;

	OctreeVertex.z = curr->origin.z - curr->halfDimension.z;
	OctreeVertex.z = OctreeVertex.z + (number & 1) * 2 * curr->halfDimension.z;

	return OctreeVertex;
}
__device__ void IteraAddPoints(Octree* curr, Myvector& testVertex) {
	printf("添加点:\n");
	if (!curr->sub) {
		float3 temp;
		temp = GetOctreeVertex(0, curr);

		float3 halfDimension = float3(curr->halfDimension);

		Push(testVertex, temp); Push(testVertex, curr->halfDimension);
		//testVertex.push_back(temp); testVertex.push_back(curr->halfDimension);
		temp = GetOctreeVertex(7, curr);
		halfDimension.x *= (-1);
		halfDimension.y *= (-1);
		halfDimension.z *= (-1);
		Push(testVertex, temp); Push(testVertex, curr->halfDimension);
		//testVertex.push_back(temp); testVertex.push_back(curr->halfDimension * (-1));
	}
	else
	{
		//if (!curr->sub) return;//判断是否存在子节点
		for (auto child : curr->children) {
			if (child->exist) IteraAddPoints(child, testVertex);

		}
	}


}
__global__ void AddPoints(Octree* curr, Myvector& testVertex) {
	IteraAddPoints(curr, testVertex);
}




void test()
{
	int ret[5] = { 1,2,3,4,5 };
	int a[5] = { 0 };
	int* dev_ret;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&dev_ret, sizeof(int) * 5);
	cudaStatus = hipMemcpy(dev_ret, ret, sizeof(int) * 5, hipMemcpyHostToDevice);

	Octree* testB;

    cudaStatus = hipMalloc((void**)&testB, sizeof(Octree));

	///Push(testVector,Mydim);
	
	AplusB <<< 1, 1 >>> (dev_ret, 10, 100,testB);

	Myvector testVector;
	hipMalloc((void**)&(testVector.buf_), 10 * sizeof(float3));
	testVector.buf_len_ = 10;//
	testVector.cnt_top_ = 0;
	//float3 Mydim;
	//Push(testVector, Mydim);

	AddPoints << < 1, 1 >> > (testB, testVector);
	//cout << testB->accuracy << endl;

	cudaStatus = hipMemcpy(a, dev_ret, sizeof(int) * 5, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	for (int i = 0; i < 5; i++)
	{
		cout << "A+B = " << a[i] << endl;
	}


	hipFree(dev_ret);
}


int main()
{
	hipEvent_t start, stop;
	float elapseTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	test();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapseTime, start, stop);
	cout << elapseTime << " ms" << endl;


	system("pause");
	return 0;

}